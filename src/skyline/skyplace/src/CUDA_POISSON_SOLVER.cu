#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <hipfft/hipfft.h>

#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "PoissonSolver.h"
#include "CUDA_DCT.h"
#include "CUDA_UTIL.h"

namespace skyplace 
{

__global__ void precomputeExpk(hipfftComplex *expkM, 
                               hipfftComplex *expkN,
                               const int M, 
                               const int N)
{
  const int tID = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(tID <= M / 2)
  {
    int hID = tID;
    hipfftComplex W_h_4M = make_float2(__cosf((float)FFT_PI * hID / (2 * M)), 
                                     -__sinf((float)FFT_PI * hID / (M * 2)));
    expkM[hID] = W_h_4M;
  }
  if(tID <= N / 2)
  {
    int wid = tID;
    hipfftComplex W_w_4N = make_float2(__cosf((float)FFT_PI * wid / (2 * N)), 
                                     -__sinf((float)FFT_PI * wid / (N * 2)));
    expkN[wid] = W_w_4N;
  }
}

__global__ void precomputeExpkForInverse(hipfftComplex *expkM, 
                                         hipfftComplex *expkN, 
                                         hipfftComplex *expkMN_1, 
                                         hipfftComplex *expkMN_2, 
                                         const int M, 
                                         const int N)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < M)
  {
    int hid = tid;
    hipfftComplex W_h_4M = make_float2(__cosf((float)FFT_PI * hid / (2 * M)), 
                                       -__sinf((float)FFT_PI * hid / (M * 2)));
    expkM[hid] = W_h_4M;
    // expkMN_1
    hipfftComplex W_h_4M_offset = make_float2(__cosf((float)FFT_PI * (hid + M) / (2 * M)), 
                                              -__sinf((float)FFT_PI * (hid + M) / (M * 2)));
    expkMN_1[hid] = W_h_4M;
    expkMN_1[hid + M] = W_h_4M_offset;
  
    //expkMN_2
    W_h_4M = make_float2(-__sinf((float)FFT_PI * (hid - (N - 1)) / (M * 2)), 
                         -__cosf((float)FFT_PI * (hid - (N - 1)) / (2 * M)));

    W_h_4M_offset = make_float2(-__sinf((float)FFT_PI * (hid - (N - 1) + M) / (M * 2)), 
                                  -__cosf((float)FFT_PI * (hid - (N - 1) + M) / (2 * M)));
    expkMN_2[hid] = W_h_4M;
    expkMN_2[hid + M] = W_h_4M_offset;
  }
  if(tid <= N / 2)
  {
    int wid = tid;
    hipfftComplex W_w_4N = make_float2(__cosf((float)FFT_PI * wid / (2 * N)), 
                                     -__sinf((float)FFT_PI * wid / (N * 2)));
    expkN[wid] = W_w_4N;
  }
}

__global__ void divideByWSquare(const int numBinX,
                                const int numBinY,
                                hipfftReal* input)
{
  //const int binID = blockDim.x * blockIdx.x + threadIdx.x;

  const int wID = blockDim.x * blockIdx.x + threadIdx.x;
  const int hID = blockDim.y * blockIdx.y + threadIdx.y;

  if(wID < numBinX && hID < numBinY)
  {
    int binID = wID + hID * numBinX;

    if(hID == 0 && wID == 0)
      input[binID] = 0.0;
    else
    {
      float denom1 = (2.0 * float(FFT_PI) * wID) / numBinX; 
      float denom2 = (2.0 * float(FFT_PI) * hID) / numBinY;

      input[binID] /= (denom1 * denom1 + denom2 * denom2);
    }
  }
}

__global__ void multiplyW(const int numBinX,
                          const int numBinY,
                          const hipfftReal* auv, 
                          hipfftReal* inputForX,
                          hipfftReal* inputForY)
{
  //const int binID = blockDim.x * blockIdx.x + threadIdx.x;
  const int wID = blockDim.x * blockIdx.x + threadIdx.x;
  const int hID = blockDim.y * blockIdx.y + threadIdx.y;

  if(wID < numBinX && hID < numBinY)
  {
    int binID = wID + hID * numBinX;

    float w_u = (2.0 * float(FFT_PI) * wID) / numBinX; 
    float w_v = (2.0 * float(FFT_PI) * hID) / numBinY; 

    inputForX[binID] = w_u * auv[binID]; // why w_v?
    inputForY[binID] = w_v * auv[binID]; // why w_u?
  }
}

PoissonSolver::PoissonSolver()
  : numBinX_            (0),
    numBinY_            (0),

    d_expkN_            (nullptr),
    d_expkM_            (nullptr),

    d_expkNForInverse_  (nullptr),
    d_expkMForInverse_  (nullptr),

    d_expkMN1_          (nullptr),
    d_expkMN2_          (nullptr),

    d_binDensity_       (nullptr),
    d_auv_              (nullptr),
    d_potential_        (nullptr),

    d_efX_              (nullptr),
    d_efY_              (nullptr),

    d_workSpaceReal1_   (nullptr),
    d_workSpaceReal2_   (nullptr),
    d_workSpaceReal3_   (nullptr),
    d_workSpaceComplex_ (nullptr),
  
    d_inputForX_        (nullptr),
    d_inputForY_        (nullptr)
{}

PoissonSolver::PoissonSolver(int numBinX, int numBinY) : PoissonSolver()
{
  numBinX_ = numBinX;
  numBinY_ = numBinY;
  init();
}

PoissonSolver::~PoissonSolver()
{
  freeDeviceMemory();
}

void
PoissonSolver::init()
{
  printf("[PoissonSolver] Start PoissonSolver Initialization!\n");

  setupForCUDAKernel();

  printf("[PoissonSolver] PoissonSolver is initialized!\n");
}

void
PoissonSolver::solvePoissonPotential(const float* binDensity,
                                           float* potential)
{
  int numThread = 16;

  dim3 gridSize( (numBinX_ + numThread - 1) / numThread, 
                 (numBinY_ + numThread - 1) / numThread, 1);

  dim3 blockSize(numThread, numThread, 1);

  // Step #1. Compute Coefficient (a_uv)
  dct_2d_fft(numBinY_, 
             numBinX_,
             plan_,
             d_expkM_,
             d_expkN_,
             binDensity, 
             d_workSpaceReal1_, 
             d_workSpaceComplex_, 
             d_auv_);

  // Step #2. Divide by (w_u^2 + w_v^2)
  divideByWSquare<<<gridSize, blockSize>>>(numBinX_, numBinY_, d_auv_);

  // Step #3. Compute Potential
  idct_2d_fft(numBinY_, 
              numBinX_,
              planInverse_,
              d_expkMForInverse_,
              d_expkNForInverse_,
              d_expkMN1_,
              d_expkMN2_,
              d_auv_, 
              d_workSpaceComplex_, 
              d_workSpaceReal1_, 
              potential);
}

void
PoissonSolver::solvePoissonForce(float* electroForceX, 
                                 float* electroForceY)
{
  int numThread = 16;

  dim3 gridSize( (numBinX_ + numThread - 1) / numThread, 
                 (numBinY_ + numThread - 1) / numThread, 1);

  dim3 blockSize(numThread, numThread, 1);

  // Step #4. Multiply w_u , w_v
  multiplyW<<<gridSize, blockSize>>>(numBinX_, 
                                     numBinY_,
                                     d_auv_, 
                                     d_inputForX_, 
                                     d_inputForY_);

  // Step #5. Compute ElectroForceX
  idxst_idct(numBinX_, 
             numBinY_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForX_, 
             d_workSpaceReal1_, 
             d_workSpaceComplex_, 
             d_workSpaceReal2_, 
             d_workSpaceReal3_, 
             electroForceX);

  // Step #6. Compute ElectroForceY
  idct_idxst(numBinY_, 
             numBinX_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForY_, 
             d_workSpaceReal1_, 
              d_workSpaceComplex_, 
             d_workSpaceReal2_, 
             d_workSpaceReal3_, 
             electroForceY);
}

void
PoissonSolver::solvePoisson(const float* binDensity,
                                  float* potential,
                                  float* electroForceX,
                                  float* electroForceY)
{
  int numThread = 16;

  dim3 gridSize( (numBinX_ + numThread - 1) / numThread, 
                 (numBinY_ + numThread - 1) / numThread, 1);

  dim3 blockSize(numThread, numThread, 1);

  // Step #1. Compute Coefficient (a_uv)
  dct_2d_fft(numBinY_, 
             numBinX_,
             plan_,
             d_expkM_,
             d_expkN_,
             binDensity, 
             d_workSpaceReal1_, 
             d_workSpaceComplex_, 
             d_auv_);

  // Step #2. Divide by (w_u^2 + w_v^2)
  divideByWSquare<<<gridSize, blockSize>>>(numBinX_, numBinY_, d_auv_);

  // Step #3. Compute Potential
  idct_2d_fft(numBinY_, 
              numBinX_,
              planInverse_,
              d_expkMForInverse_,
              d_expkNForInverse_,
              d_expkMN1_,
              d_expkMN2_,
              d_auv_, 
              d_workSpaceComplex_, 
              d_workSpaceReal1_, 
              potential);

  // Step #4. Multiply w_u , w_v
  multiplyW<<<gridSize, blockSize>>>(numBinX_, 
                                     numBinY_,
                                     d_auv_, 
                                     d_inputForX_, 
                                     d_inputForY_);

  // Step #5. Compute ElectroForceX
  idxst_idct(numBinY_, 
             numBinX_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForX_, 
             d_workSpaceReal1_, 
              d_workSpaceComplex_, 
             d_workSpaceReal2_, 
             d_workSpaceReal3_, 
             electroForceX);

  // Step #6. Compute ElectroForceY
  idct_idxst(numBinY_, 
             numBinX_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForY_, 
             d_workSpaceReal1_, 
             d_workSpaceComplex_, 
             d_workSpaceReal2_, 
             d_workSpaceReal3_, 
             electroForceY);

  hipDeviceSynchronize();
}

void
PoissonSolver::setupForCUDAKernel()
{
  printf("[CUDA-POISSON] Setup for CUDA Kernel.\n");

  CUDA_CHECK(hipMalloc((void**)&d_binDensity_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_auv_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_potential_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_efX_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_efY_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_workSpaceReal1_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_workSpaceReal2_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_workSpaceReal3_,
                         numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**)&d_workSpaceComplex_,
                         (numBinX_ / 2 + 1) * numBinY_ * sizeof(hipfftComplex)));

  // expk
  // For DCT2D
  CUDA_CHECK(hipMalloc((void **)&d_expkM_,  
                        (numBinY_ / 2 + 1) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void **)&d_expkN_,  
                        (numBinX_ / 2 + 1) * sizeof(hipfftComplex)));

  // For IDCT2D & IDXST_IDCT & IDCT_IDXST
  CUDA_CHECK(hipMalloc((void **)&d_expkMForInverse_,  
                        (numBinY_) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void **)&d_expkNForInverse_,  
                        (numBinX_ / 2 + 1) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void **)&d_expkMN1_,  
                        (numBinX_ + numBinY_) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void **)&d_expkMN2_,  
                        (numBinX_ + numBinY_) * sizeof(hipfftComplex)));

  // For Input For IDXST_IDCT & IDCT_IDXST
  CUDA_CHECK(hipMalloc((void **)&d_inputForX_,  
                        numBinX_ * numBinY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void **)&d_inputForY_,  
                        numBinX_ * numBinY_ * sizeof(hipfftReal)));

  int numThread = 1024;
  int numBin = std::max(numBinX_, numBinY_);
  int numBlock  = (numBin - 1 + numThread) / numThread;

  precomputeExpk<<<numBlock, numThread>>>(d_expkM_, 
                                          d_expkN_, 
                                          numBinY_, 
                                          numBinX_);

  precomputeExpkForInverse<<<numBlock, numThread>>>(d_expkMForInverse_, 
                                                    d_expkNForInverse_,
                                                    d_expkMN1_, 
                                                    d_expkMN2_,
                                                    numBinY_, 
                                                    numBinX_);

  hipfftPlan2d(&plan_,        numBinY_, numBinX_, HIPFFT_R2C);
  hipfftPlan2d(&planInverse_, numBinY_, numBinX_, HIPFFT_C2R);

  printf("[CUDA-POISSON] Finish Setup.\n");
}

void
PoissonSolver::freeDeviceMemory()
{
  CUDA_CHECK(hipFree(d_binDensity_));
  CUDA_CHECK(hipFree(d_auv_));
  CUDA_CHECK(hipFree(d_potential_));

  CUDA_CHECK(hipFree(d_efX_));
  CUDA_CHECK(hipFree(d_efY_));

  CUDA_CHECK(hipFree(d_workSpaceReal1_));
  CUDA_CHECK(hipFree(d_workSpaceReal2_));
  CUDA_CHECK(hipFree(d_workSpaceReal3_));

  CUDA_CHECK(hipFree(d_workSpaceComplex_));

  CUDA_CHECK(hipFree(d_expkN_));
  CUDA_CHECK(hipFree(d_expkM_));

  CUDA_CHECK(hipFree(d_expkNForInverse_));
  CUDA_CHECK(hipFree(d_expkMForInverse_));

  CUDA_CHECK(hipFree(d_expkMN1_));
  CUDA_CHECK(hipFree(d_expkMN2_));

  CUDA_CHECK(hipFree(d_inputForX_));
  CUDA_CHECK(hipFree(d_inputForY_));

  hipfftDestroy(plan_);
  hipfftDestroy(planInverse_);
}

} // namespace skyplace
