#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <vector>
#include <iostream>
#include <thrust/partition.h>
#include <fstream>
#include <sstream>

namespace Louvain
{

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures,
          aggregation_phase_structures& aggregationPhaseStructures) {
  // copying from deviceStructures to hostStructures
  int V = hostStructures.V, E = hostStructures.E;
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&deviceStructures.toOwnCommunity, V * sizeof(int)));

  thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + V, 1);
  thrust::sequence(thrust::device, deviceStructures.vertexCommunity, deviceStructures.vertexCommunity + V, 0);
  thrust::sequence(thrust::device, deviceStructures.newVertexCommunity, deviceStructures.newVertexCommunity + V, 0);
  thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

  HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));

  // preparing aggregationPhaseStructures
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.communityDegree, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newID, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgePos, V * sizeof(int)));;
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.vertexStart, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.orderedVertices, V * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgeIndexToCurPos, E * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newEdges, E * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newWeights, E * sizeof(float)));
}

void deleteStructures(device_structures& deviceStructures,
           aggregation_phase_structures& aggregationPhaseStructures) 
{
  HANDLE_ERROR(hipFree(deviceStructures.originalV));
  HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
  HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
  HANDLE_ERROR(hipFree(deviceStructures.edges));
  HANDLE_ERROR(hipFree(deviceStructures.weights));
  HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
  HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
  HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
  HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
  HANDLE_ERROR(hipFree(deviceStructures.E));
  HANDLE_ERROR(hipFree(deviceStructures.V));
  HANDLE_ERROR(hipFree(deviceStructures.communitySize));
  HANDLE_ERROR(hipFree(deviceStructures.partition));
  HANDLE_ERROR(hipFree(deviceStructures.toOwnCommunity));

  HANDLE_ERROR(hipFree(aggregationPhaseStructures.communityDegree));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.newID));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgePos));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.vertexStart));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.orderedVertices));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgeIndexToCurPos));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.newEdges));
  HANDLE_ERROR(hipFree(aggregationPhaseStructures.newWeights));
}

int blocksNumber(int V, int threadsPerVertex) 
{
  return (V * threadsPerVertex + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

bool isPrime(int n) 
{
  for (int i = 2; i < sqrt(n) + 1; i++)
    if (n % i == 0)
      return false;
  return true;
}

int getPrime(int n) 
{
  do {
    n++;
  } while(!isPrime(n));
  return n;
}

};
